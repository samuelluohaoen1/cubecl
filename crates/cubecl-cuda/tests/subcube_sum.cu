
#include <hip/hip_runtime.h>
typedef unsigned int uint;

extern "C" __global__ void kernel(float output_0[], uint info[]) {

  int threadIdxGlobal = threadIdx.x + threadIdx.y * blockDim.x +
                        threadIdx.z * (blockDim.x * blockDim.y);

  int warpSizeChecked = min(warpSize, blockDim.x * blockDim.y * blockDim.z);
  uint rank = info[0];
  uint rank_2 = rank * 2;
  float l_0_0;
  float l_0_1;
  bool l_0_2;
  uint l_0_3;
  bool l_0_4;
  l_0_3 = info[(1 * 2 * info[0]) + 1];
  l_0_4 = threadIdxGlobal < l_0_3;
  if (l_0_4) {
    l_0_0 = output_0[threadIdxGlobal];
  } else {
    l_0_0 = float(0.0);
  }

  l_0_1 = l_0_0;
  {
    for (int offset = warpSizeChecked / 2; offset > 0; offset /= 2) {
      l_0_1 += __shfl_down_sync(0xFFFFFFFF, l_0_1, offset);
    }
  }
  l_0_2 = threadIdxGlobal == uint(0);
  if (l_0_2) {
    uint l_1_0;
    bool l_1_1;
    l_1_0 = info[(1 * 2 * info[0]) + 1];
    l_1_1 = uint(0) < l_1_0;
    if (l_1_1) {
      output_0[uint(0)] = l_0_1;
    }
  }
}